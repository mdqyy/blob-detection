#include "hip/hip_runtime.h"
#include <cutil.h>
#include <stdio.h>
#include <stdlib.h>

#define THRESH 70000
#define MAX(a, b) (((a) > (b)) ? (a) : (b))
#define MIN(a, b) (((a) < (b)) ? (a) : (b))

#define RADIUS 1
#define BLOCKSIZE 16

#define ROWS (BLOCKSIZE + 2 * RADIUS)
#define COLS (BLOCKSIZE + 2 * RADIUS)

#define DIM (ROWS * COLS)

#define WIDTH (BLOCKSIZE + 2)
typedef char guchar;

/*__shared__ guchar sdevice_image[ROWS][COLS];
__shared__ guchar sdevice_x[ROWS][COLS];
__shared__ guchar sdevice_y[ROWS][COLS];

__shared__ float sxx_grad[ROWS][COLS];
__shared__ float sxy_grad[ROWS][COLS];
__shared__ float syy_grad[ROWS][COLS];

__shared__ float sharris[ROWS][COLS];
*/

__shared__ guchar sdevice_image[DIM];
__shared__ guchar sdevice_x[DIM];
__shared__ guchar sdevice_y[DIM];

__shared__ float sxx_grad[DIM];
__shared__ float sxy_grad[DIM];
__shared__ float syy_grad[DIM];

__shared__ float sharris[DIM];

__device__ void gpu_sobel_shared_op(int y, int x);
__device__ void gpu_grad_matrix_shared_op(int y, int x);
__device__ void gpu_blur_shared_op(int y, int x, int rstart, int rend, int cstart, int cend, float device_image[DIM]);
__device__ void gpu_harris_shared_op(int y, int x);
__device__ void gpu_nonmax_shared_op(int y, int x, int rstart, int rend, int cstart, int cend);
__device__ void gpu_threshold_shared_op(int y, int x, const int index, guchar *device_image);

__global__ void kernel_shared_op(guchar *device_image, int width, int height)
{
	const int ix = __mul24(blockDim.x, blockIdx.x) + threadIdx.x;
	const int iy = __mul24(blockDim.y, blockIdx.y) + threadIdx.y;

	const int index = __mul24(iy, width) + ix;

	const int x = threadIdx.x + RADIUS;
	const int y = threadIdx.y + RADIUS;

	if(threadIdx.x == 0 && threadIdx.y == 0)
	{
		sdevice_image[(y-RADIUS) * WIDTH + x-RADIUS] = device_image[(iy-RADIUS) * width + (ix-RADIUS)];
		sdevice_image[(y-RADIUS) * WIDTH + x+BLOCKSIZE] = device_image[(iy-RADIUS) * width + (ix+BLOCKSIZE)];
		sdevice_image[(y+BLOCKSIZE) * WIDTH + x-RADIUS] = device_image[(iy+BLOCKSIZE) * width + (ix-RADIUS)];
		sdevice_image[(y+BLOCKSIZE) * WIDTH + x+BLOCKSIZE] = device_image[(iy+BLOCKSIZE) * width + (ix+BLOCKSIZE)];		
	}
	if(threadIdx.x < BLOCKSIZE && threadIdx.y == 0)
	{
		sdevice_image[(y-RADIUS) * WIDTH + x] = device_image[(iy-RADIUS) * width + ix];
		sdevice_image[(y+BLOCKSIZE) * WIDTH + x] = device_image[(iy+BLOCKSIZE) * width + ix];
	}
	if(threadIdx.x == 0 && threadIdx.y < BLOCKSIZE)
	{
		sdevice_image[y * WIDTH + x-RADIUS] = device_image[iy * width + (ix-RADIUS)];
		sdevice_image[y * WIDTH + x+BLOCKSIZE] = device_image[iy * width + (ix+BLOCKSIZE)];
	}

	sdevice_image[y * WIDTH + x] = device_image[index];

	__syncthreads();

	gpu_sobel_shared_op(y, x);

	__syncthreads();

	gpu_grad_matrix_shared_op(y, x);

	//__syncthreads();
	/*int rstart = MAX(RADIUS, y-1);
	int rend = MIN(BLOCKSIZE, y+1);

	int cstart = MAX(RADIUS, x-1);
	int cend = MIN(BLOCKSIZE, x+1);
	*/

	int rstart = max(RADIUS, y-1);
	int rend = min(BLOCKSIZE, y+1);

	int cstart = max(RADIUS, x-1);
	int cend = min(BLOCKSIZE, x+1);

	gpu_blur_shared_op(y, x, rstart, rend, cstart, cend, sxx_grad);
	gpu_blur_shared_op(y, x, rstart, rend, cstart, cend, sxy_grad);
	gpu_blur_shared_op(y, x, rstart, rend, cstart, cend, syy_grad);

	//__syncthreads();

	gpu_harris_shared_op(y, x);

	//__syncthreads();

    gpu_nonmax_shared_op(y, x, rstart, rend, cstart, cend);

	//__syncthreads();

    gpu_threshold_shared_op(y, x, index, device_image);
}


__device__ void gpu_sobel_shared_op(int y, int x)
{
	/*int sumx = (-sdevice_image[y-1][x-1]) + (sdevice_image[y-1][x+1]) + 
			   (__mul24(-2, sdevice_image[y][x-1])) + (__mul24(2, sdevice_image[y][x+1])) + 
			   (-sdevice_image[y+1][x-1]) + (sdevice_image[y+1][x+1]);

	int sumy = (-sdevice_image[y-1][x-1]) + (__mul24(-2, sdevice_image[y-1][x])) + 
			   (-sdevice_image[y-1][x+1]) + (sdevice_image[y+1][x-1]) + 
			   (__mul24(2, sdevice_image[y+1][x])) + (sdevice_image[y+1][x+1]);
	  */
	sdevice_x[y * WIDTH + x] = (-sdevice_image[(y-1) * WIDTH + x-1]) + (sdevice_image[(y-1) * WIDTH + x+1]) + 
			   		  (__mul24(-2, sdevice_image[y * WIDTH + x-1])) + (__mul24(2, sdevice_image[y * WIDTH + x+1])) + 
			   		  (-sdevice_image[(y+1) * WIDTH + x-1]) + (sdevice_image[(y+1) * WIDTH + x+1]);
	sdevice_y[y * WIDTH + x] = (-sdevice_image[(y-1) * WIDTH + x-1]) + (__mul24(-2, sdevice_image[(y-1) * WIDTH + x])) + 
			   	  	  (-sdevice_image[(y-1) * WIDTH + x+1]) + (sdevice_image[(y+1) * WIDTH + x-1]) + 
			   		  (__mul24(2, sdevice_image[(y+1) * WIDTH + x])) + (sdevice_image[(y+1) * WIDTH + x+1]);
}


__device__ void gpu_blur_shared_op(int y, int x, int rstart, int rend, int cstart, int cend, float device_image[DIM])
{
	int r, c;

	int sum = 0;
	int n = 0;
	
	/*int rstart = MAX(RADIUS, y-1);
	int rend = MIN(BLOCKSIZE, y+1);

	int cstart = MAX(RADIUS, x-1);
	int cend = MIN(BLOCKSIZE, x+1);
	*/

	for(r = rstart; r <= rend; r++)
	{
		int row = r * WIDTH;
		for(c = cstart; c <= cend; c++)
		{
			sum += device_image[row + c];
			n++;
		}
	}

	device_image[y * WIDTH + x] =  __fdividef(sum, n);
}

__device__ void gpu_grad_matrix_shared_op(int y, int x)
{
	int index = y * WIDTH + x;

	sxx_grad[index] = __mul24(sdevice_x[index], sdevice_x[index]);
	sxy_grad[index] = __mul24(sdevice_x[index], sdevice_y[index]);
	syy_grad[index] = __mul24(sdevice_y[index], sdevice_y[index]);
}

__device__ void gpu_harris_shared_op(int y, int x)
{
	int index = y * WIDTH + x;

	float det = __mul24(sxx_grad[index], syy_grad[index]) - __mul24(sxy_grad[index], sxy_grad[index]);
	float trace = sxx_grad[index] + syy_grad[index];

	sharris[index] = det - 0.06 * __powf(trace, 2);
}

__device__ void gpu_nonmax_shared_op(int y, int x, int rstart, int rend, int cstart, int cend)
{
	int r, c;
	
	float curr = sharris[y * WIDTH + x];
	
	for(r = rstart; r <= rend; r++)
	{
		int row = r * WIDTH;
		for(c = cstart; c <= cend; c++)
		{
			if(curr > sharris[row + c])
			{
				sharris[row + c] = 0;
			}	
			else
			{	
				curr = sharris[row + c];
			}

		}
	}

}

__device__ void gpu_threshold_shared_op(int y, int x, const int index, guchar *device_image)
{
	device_image[index] = sharris[y * WIDTH + x] > THRESH ? (guchar)255 : device_image[index]; 
}

